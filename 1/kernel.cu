#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
// Size of array



// Kernel
__device__ double globalArr[4][3];


__global__ void Kernel() {
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 3; j++) {
			printf("%f ", globalArr[i][j]);
		}
		printf("\n");
	}
}
// Main program
int main()
{
	
	size_t bytes = 3 * 4 * sizeof(double);
	double TwoDim[3][4];

	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 4; j++) {
			TwoDim[i][j] = rand() % 10 + 1;
		}
	}

	double TwoDimD[4][3];

	hipMemcpy(globalArr, TwoDim, bytes,hipMemcpyHostToDevice);
	
	

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Launch kernel
	
	Kernel << < 1, 1 >> > ();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); // Sync events (BLOCKS till last (stop in this case) has been recorded!)
	float elapsedTime; // Initialize elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	
	return 0;
}