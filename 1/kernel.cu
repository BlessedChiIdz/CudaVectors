#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#define BLOCK_DIM 1
__shared__ float temp[BLOCK_DIM][BLOCK_DIM];

__global__ void transposeMatrixFast(float* inputMatrix, float* outputMatrix, int width, int height)
{
	

	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	if ((xIndex < width) && (yIndex < height))
	{
		int idx = yIndex * width + xIndex;

		temp[threadIdx.y][threadIdx.x] = inputMatrix[idx];
	}

	__syncthreads();

	xIndex = blockIdx.y * blockDim.y + threadIdx.x;
	yIndex = blockIdx.x * blockDim.x + threadIdx.y;

	if ((xIndex < height) && (yIndex < width))
	{
		int idx = yIndex * height + xIndex;

		outputMatrix[idx] = temp[threadIdx.x][threadIdx.y];
	}
}

__host__ void printMatrixToFile(char* fileName, float* matrix, int width, int height)
{
    FILE* file = fopen(fileName, "wt");
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            fprintf(file, "%f\t", matrix[y * width + x]);
        }
        fprintf(file, "\n");
    }
    fclose(file);
}




int main()
{
    int width = 3;   
    int height = 4;

    int matrixSize = width * height;
    int byteSize = matrixSize * sizeof(float);

    float* inputMatrix = new float[matrixSize];
    float* outputMatrix = new float[matrixSize];

    for (int i = 0; i < matrixSize; i++)
    {
        inputMatrix[i] = i;
    }
    int qwe = 0;
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            printf("%f ", inputMatrix[qwe]);
            qwe++;
        }
        printf("\n");
    }
    
    
        float* devInputMatrix;
        float* devOutputMatrix;

        hipMalloc((void**)&devInputMatrix, byteSize);
        hipMalloc((void**)&devOutputMatrix, byteSize);

        hipMemcpy(devInputMatrix, inputMatrix, byteSize, hipMemcpyHostToDevice);

        dim3 gridSize = dim3(width / BLOCK_DIM, height / BLOCK_DIM, 1);
        dim3 blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);

        hipEvent_t start;
        hipEvent_t stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);

       
        
        transposeMatrixFast << <gridSize, blockSize >> > (devInputMatrix, devOutputMatrix, width, height);

        hipEventRecord(stop, 0);

        float time = 0;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        printf("GPU compute time: %.0f\n", time);

        hipMemcpy(outputMatrix, devOutputMatrix, byteSize, hipMemcpyDeviceToHost);

        hipFree(devInputMatrix);
        hipFree(devOutputMatrix);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    

    printMatrixToFile("after.txt", outputMatrix, height, width);

    delete[] inputMatrix;
    delete[] outputMatrix;

    return 0;
}